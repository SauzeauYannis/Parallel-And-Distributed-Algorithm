#include "hip/hip_runtime.h"
#include <iostream>
#include <exo4/student.h>
#include <OPP_cuda.cuh>

using uchar = unsigned char;

namespace 
{
	// idem exo3
	template<int NB_WARPS>
	__device__ 
	__forceinline__
	void loadSharedMemoryCommutative(float const*const data) 
	{
		float*const shared = OPP::CUDA::getSharedMemory<float>();
		float sum = 0.f;
		const unsigned globalOffset = blockIdx.x * 1024; 
		for(auto tid = threadIdx.x; tid < 1024; tid += 32*NB_WARPS) 
			sum += data[tid + globalOffset];
		shared[threadIdx.x] = sum;
		__syncthreads();
	}

	// nouvelle version :-)
	__device__ 
	__forceinline__
	void reduceJumpingStep(const int jump)
	{
		float *const shared = OPP::CUDA::getSharedMemory<float>();
		const auto tid = threadIdx.x;
		if (tid < jump) 
			shared[tid] += shared[tid + jump]; 
		__syncthreads();
	}

	// similaire précédente, mais boucle différente (les threads qui travaillent sont en tête ...)
	template<int NB_WARPS>
	__device__
	__forceinline__
	float reducePerBlock(
		float const*const source
	) {
		float*const shared = OPP::CUDA::getSharedMemory<float>();
		loadSharedMemoryCommutative<NB_WARPS>(source);
		for (int i= 32 * NB_WARPS / 2; i > 0; i >>= 1) 
			reduceJumpingStep(i);
		return shared[0]; 
	}

	
	// idem exo3
	template<int NB_WARPS>
	__device__
	__forceinline__
	void fillBlock(
		const float color, 
		float*const result
	) {
		const auto offset = blockIdx.x * 1024;
		unsigned tid = threadIdx.x;

		while(tid < 1024) {
				result[tid + offset] = color;
				tid += 32 * NB_WARPS;
		}
	}


	// idem exo1
	template<int NB_WARPS>
	struct EvaluateWarpNumber {
		enum { res = 1 };
	};
	template<>
	struct EvaluateWarpNumber<1> {
		enum { res = 16 };
	};
	template<>
	struct EvaluateWarpNumber<2> {
		enum { res = 8 };
	};
	template<>
	struct EvaluateWarpNumber<4> {
		enum { res = 4 };
	};
	template<>
	struct EvaluateWarpNumber<8> {
		enum { res = 4 };
	};
	template<>
	struct EvaluateWarpNumber<16> {
		enum { res = 2 };
	};
	template<int NB_WARPS=32>
	__global__
	__launch_bounds__(32*NB_WARPS , EvaluateWarpNumber<NB_WARPS>::res)
	void blockEffectKernel( 
		float const*const source, 
		float *const result
	) {
		const float sumInBlock = reducePerBlock<NB_WARPS>(source);
		fillBlock<NB_WARPS>(sumInBlock, result);
	}
}


// Attention : ici la taille des vecteurs n'est pas toujours un multiple du nombre de threads !
// Il faut donc corriger l'exemple du cours ...
void StudentWorkImpl::run_blockEffect(
	OPP::CUDA::DeviceBuffer<float>& dev_source,
	OPP::CUDA::DeviceBuffer<float>& dev_result,
	const unsigned nbWarps
) {
	const auto size = dev_source.getNbElements();
	dim3 threads( 32 * nbWarps );
	dim3 blocks( (size + 1023) / 1024 );
	const size_t sizeSharedMemory(threads.x*sizeof(float));
	switch(nbWarps) {
		case 1:
			::blockEffectKernel<1> <<<blocks, threads, sizeSharedMemory>>>(
				dev_source.getDevicePointer(),
				dev_result.getDevicePointer()
			);
			return;
		case 2:
			::blockEffectKernel<2> <<<blocks, threads, sizeSharedMemory>>>(
				dev_source.getDevicePointer(),
				dev_result.getDevicePointer()
			);
			return;
		case 4:
			::blockEffectKernel<4> <<<blocks, threads, sizeSharedMemory>>>(
				dev_source.getDevicePointer(),
				dev_result.getDevicePointer()
			);
			return;
		case 8:
			::blockEffectKernel<8> <<<blocks, threads, sizeSharedMemory>>>(
				dev_source.getDevicePointer(),
				dev_result.getDevicePointer()
			);
			return;
		case 16:
			::blockEffectKernel<16> <<<blocks, threads, sizeSharedMemory>>>(
				dev_source.getDevicePointer(),
				dev_result.getDevicePointer()
			);
			return;
		case 32:
			::blockEffectKernel<32><<<blocks, threads, sizeSharedMemory>>>(
				dev_source.getDevicePointer(),
				dev_result.getDevicePointer()
			);
			return;
		default:
			::blockEffectKernel<32><<<blocks, threads, sizeSharedMemory>>>(
				dev_source.getDevicePointer(),
				dev_result.getDevicePointer()
			);
			return;
	}

}