#include "hip/hip_runtime.h"
#include <iostream>
#include <exo2/student.h>
#include <OPP_cuda.cuh>

using uchar = unsigned char;

namespace 
{
	// L'opération est associative (enfin, en toute généralité), et donc les permutations de valeurs sont interdites.
	// Seul les changements de parenthèses sont autorisées ...
	// Donc il y a deux solutions :
	// - La plus simple est d'effectuer plusieurs réductions successives par blocs
	// - La plus difficile mais efficace, et de grouper les valeurs consécutives par thread.
	// Avec cette seconde, le premier thread (0) va traiter des valeurs consécutives. Le thread suivant aussi, etc.
	// En supposant par exemple que chaque thread traite 4 valeurs, alors les 4 premiers pixels du blocs sont utilisés par
	// le thread 0, le 4 suivant par le thread 1, etc. jusqu'au thread 255 ;-)
	// NB : on suppose que le nombre de warps est une puissance de 2 (et donc divise 1024)
	template<int NB_WARPS>
	__device__ 
	__forceinline__
	void loadSharedMemoryAssociate(float const*const data) 
	{
		float*const shared = OPP::CUDA::getSharedMemory<float>();

		const auto globalOffset = 1024 * blockIdx.x;
		const auto localThreadId = threadIdx.x;
		const unsigned nbPixelsPerThread = (1024 + 32*NB_WARPS - 1) / (32*NB_WARPS);

		float sumPerThread = 0.f;

		for(unsigned i=0; i < nbPixelsPerThread; ++i) 
		{
			// indice du pixel à traiter
			const auto pixelIdInBlock = nbPixelsPerThread * localThreadId + i;
			
			sumPerThread += data[pixelIdInBlock + globalOffset];
		}
		shared[localThreadId] = sumPerThread;
		__syncthreads();
	}


	// idem exo1, sauf test de débordement
	__device__ 
	__forceinline__
	void reduceJumpingStep(const int jump)
	{
		//TODO TEST DE DEBORDEMENT
		float *const shared = OPP::CUDA::getSharedMemory<float>();
		const auto tid = threadIdx.x;
		if((tid % (jump<<1)) == 0) 
			shared[tid] += shared[tid+jump]; 
		__syncthreads();
	}


	// on ne changera ici que le nombre d'itérations (10 avant, ici moins)
	template<int NB_WARPS>
	__device__
	__forceinline__
	float reducePerBlock(
		float const*const source
	) {
		float*const shared = OPP::CUDA::getSharedMemory<float>();
		loadSharedMemoryAssociate<NB_WARPS>(source);
		for(int i=1; i<32*NB_WARPS; i<<=1) 
			reduceJumpingStep(i);
		return shared[0];
	}	
	

	// ressemble beaucoup à l'exo1 ...
	template<int NB_WARPS>
	__device__
	__forceinline__
	void fillBlock(
		const float color, 
		float*const result
	) {
		// calcul de l'offset du bloc : la taille est 1024
		const auto offset = blockIdx.x * 1024;
		
		unsigned tid = threadIdx.x;

		while (tid < 1024) {
				result[tid + offset] = color;
				tid += 32 * NB_WARPS;
		}
	}


	// idem exo1 with templates
	template<int NB_WARPS>
	struct EvaluateWarpNumber {
		enum { res = 1 };
	};
	template<>
	struct EvaluateWarpNumber<1> {
		enum { res = 16 };
	};
	template<>
	struct EvaluateWarpNumber<2> {
		enum { res = 8 };
	};
	template<>
	struct EvaluateWarpNumber<4> {
		enum { res = 4 };
	};
	template<>
	struct EvaluateWarpNumber<8> {
		enum { res = 4 };
	};
	template<>
	struct EvaluateWarpNumber<16> {
		enum { res = 2 };
	};

	// idem exo1
	template<int NB_WARPS=32>
	__global__
	__launch_bounds__(32*NB_WARPS , EvaluateWarpNumber<NB_WARPS>::res)
	void blockEffectKernel( 
		float const*const source, 
		float *const result
	) {
		const float sumInBlock = reducePerBlock<NB_WARPS>(source);
		fillBlock<NB_WARPS>(sumInBlock, result);
	}
}


// idem exo1, sauf la taille d'un bloc de threads (et les templates)
void StudentWorkImpl::run_blockEffect(
	OPP::CUDA::DeviceBuffer<float>& dev_source,
	OPP::CUDA::DeviceBuffer<float>& dev_result,
	const unsigned nbWarps
) {
	// Le nombre de warps est réduit ...
	const auto size = dev_source.getNbElements();
	// Le nombre de threads par bloc dépend du nombre de warps ;-)
	dim3 threads(32 * nbWarps); 
	// Attention : le nombre de blocs est calculer en considérant des traitements de 1024 pixels ! 
	dim3 blocks ((size + 1024-1) / 1024 );
	// le reste est classique
	const size_t sizeSharedMemory(threads.x*sizeof(float));
	switch(nbWarps) {
		case 1:
			::blockEffectKernel<1> <<<blocks, threads, sizeSharedMemory>>>(
				dev_source.getDevicePointer(),
				dev_result.getDevicePointer()
			);
			return;
		case 2:
			::blockEffectKernel<2> <<<blocks, threads, sizeSharedMemory>>>(
				dev_source.getDevicePointer(),
				dev_result.getDevicePointer()
			);
			return;
		case 4:
			::blockEffectKernel<4> <<<blocks, threads, sizeSharedMemory>>>(
				dev_source.getDevicePointer(),
				dev_result.getDevicePointer()
			);
			return;
		case 8:
			::blockEffectKernel<8> <<<blocks, threads, sizeSharedMemory>>>(
				dev_source.getDevicePointer(),
				dev_result.getDevicePointer()
			);
			return;
		case 16:
			::blockEffectKernel<16> <<<blocks, threads, sizeSharedMemory>>>(
				dev_source.getDevicePointer(),
				dev_result.getDevicePointer()
			);
			return;
		case 32:
			::blockEffectKernel<32><<<blocks, threads, sizeSharedMemory>>>(
				dev_source.getDevicePointer(),
				dev_result.getDevicePointer()
			);
			return;
		default:
			::blockEffectKernel<32><<<blocks, threads, sizeSharedMemory>>>(
				dev_source.getDevicePointer(),
				dev_result.getDevicePointer()
			);
			return;
	}
}