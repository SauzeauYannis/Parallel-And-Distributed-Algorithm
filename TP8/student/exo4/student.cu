#include "hip/hip_runtime.h"
#include <iostream>
#include <exo4/student.h>
#include <OPP_cuda.cuh>

namespace 
{
  using uchar = unsigned char;
 
	__global__
  void transformation_kernel(
			const float* const value,
      const unsigned* const repartition,
      float* const transformation,
      const unsigned size
  ){
		const unsigned tid = blockIdx.x * blockDim.x + threadIdx.x;
            
		if (tid < size) {
      const uchar xi = uchar(value[tid] * 256.f);
      transformation[tid] = (255.f * float(repartition[xi])) / (256.f * size);
    }
  }
}

bool StudentWorkImpl::isImplemented() const {
	return true;
}

void StudentWorkImpl::run_Transformation(
	OPP::CUDA::DeviceBuffer<float>& dev_Value,
	OPP::CUDA::DeviceBuffer<unsigned>& dev_repartition,
	OPP::CUDA::DeviceBuffer<float>& dev_transformation // or "transformed"
) {
  //const unsigned nbThreads = 32;
  //const unsigned nbThreads = 64;
  //const unsigned nbThreads = 128;
  //const unsigned nbThreads = 256;
  //const unsigned nbThreads = 512;
  const unsigned nbThreads = 1024;
   
  const unsigned size = dev_Value.getNbElements();
   
	const dim3 threads(nbThreads);
	const dim3 blocks((size + nbThreads - 1) / nbThreads);

  transformation_kernel<<<blocks,threads>>>(
      dev_Value.getDevicePointer(),
      dev_repartition.getDevicePointer(),
      dev_transformation.getDevicePointer(),
      size
	);
}
